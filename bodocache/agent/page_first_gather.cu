
#include <hip/hip_runtime.h>
// Stub CUDA kernel placeholder for page-first layout gather/scatter.
extern "C" __global__ void page_first_gather(const float* __restrict__ src,
                                             float* __restrict__ dst,
                                             int stride_src, int stride_dst,
                                             int page_elems) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < page_elems) {
    dst[idx] = src[idx];
  }
}

