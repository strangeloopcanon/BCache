#include "copy_engine_common.hpp"

#ifdef USE_CUDA_BACKEND

#include <hip/hip_runtime.h>

struct CudaBackend {
  using stream_t = hipStream_t;
  std::vector<std::vector<stream_t>> streams_; // [device][stream_id]

  void init_device_streams(int device, int streams_per_dev) {
    int device_count = 0;
    hipGetDeviceCount(&device_count);
    if (device < 0 || device >= device_count) {
      throw std::runtime_error("invalid CUDA device id");
    }
    hipSetDevice(device);
    streams_.resize(device + 1);
    auto& vec = streams_[device];
    vec.resize(streams_per_dev);
    for (int i = 0; i < streams_per_dev; ++i) {
      hipStreamCreateWithFlags(&vec[i], hipStreamNonBlocking);
    }
  }

  stream_t get_stream(int device, int stream_id) {
    if (stream_id < 0) stream_id = 0;
    auto& vec = streams_.at(device);
    if (vec.empty()) throw std::runtime_error("streams not initialized");
    return vec[stream_id % static_cast<int>(vec.size())];
  }

  void* alloc_pinned(size_t bytes) {
    void* p = nullptr;
    hipError_t st = hipHostAlloc(&p, bytes, hipHostMallocDefault);
    if (st != hipSuccess) return nullptr;
    return p;
  }

  void free_pinned(void* p) { hipHostFree(p); }

  void memcpy_h2d_async(int device, void* dst_device, const void* src_host, size_t bytes, stream_t s) {
    hipSetDevice(device);
    hipMemcpyAsync(dst_device, src_host, bytes, hipMemcpyHostToDevice, s);
  }

  void record_event(stream_t s, void** out_event) {
    hipEvent_t ev;
    hipEventCreateWithFlags(&ev, hipEventDisableTiming);
    hipEventRecord(ev, s);
    *out_event = reinterpret_cast<void*>(ev);
  }

  bool event_completed(void* event) {
    hipEvent_t ev = reinterpret_cast<hipEvent_t>(event);
    hipError_t q = hipEventQuery(ev);
    return q == hipSuccess;
  }

  void destroy_event(void* event) {
    hipEvent_t ev = reinterpret_cast<hipEvent_t>(event);
    hipEventDestroy(ev);
  }
};

using CopyEngineCuda = CopyEngineNative<CudaBackend>;

PYBIND11_MODULE(bodocache_agent_copy_engine, m) {
  py::class_<CopyEngineCuda>(m, "CopyEngine")
      .def(py::init<int, int>(), py::arg("device_id") = 0, py::arg("streams_per_device") = 4)
      .def("acquire_host_buffer", &CopyEngineCuda::acquire_host_buffer, py::arg("bytes"))
      .def("submit", &CopyEngineCuda::submit, py::arg("ops"), py::arg("callback"));
}

#endif // USE_CUDA_BACKEND

